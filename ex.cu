#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>

// Função do kernel CUDA para somar dois vetores
__global__ void addVectors(int *a, int *b, int *c, int size) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size) {
        c[tid] = a[tid] + b[tid];
    }
}

// Função que realiza a chamada da função CUDA
void performCUDAOperation(int *d_a, int *d_b, int *d_c, int size) {
    // Configurar a grade de threads e os blocos
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;

    // Chamar o kernel CUDA para somar os vetores
    addVectors<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, size);

    // Sincronizar o dispositivo para garantir que o kernel seja concluído
    hipDeviceSynchronize();

    // Copiar o resultado de volta para o host (aqui você pode realizar operações adicionais se necessário)
    // Exemplo: cudaMemcpy(c, d_c, size * sizeof(int), cudaMemcpyDeviceToHost);
}

int main() {
    const int size = 10; // Tamanho dos vetores
    const int iterations = 5; // Número de iterações do loop
    int a[size], b[size], c[size]; // Vetores de entrada e saída no host
    int *d_a, *d_b, *d_c; // Vetores no device (GPU)

    // Inicialização dos vetores no host
    for (int i = 0; i < size; ++i) {
        a[i] = i;
        b[i] = 2 * i;
    }

    // Alocação de memória no device
    hipMalloc((void **)&d_a, size * sizeof(int));
    hipMalloc((void **)&d_b, size * sizeof(int));
    hipMalloc((void **)&d_c, size * sizeof(int));

    // Copiar dados do host para o device
    hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    // Loop para realizar várias iterações
    for (int iter = 0; iter < iterations; ++iter) {
        // Chamar a função que executa a operação CUDA
        performCUDAOperation(d_a, d_b, d_c, size);

        // Copiar os resultados de volta para o host
        hipMemcpy(c, d_c, size * sizeof(int), hipMemcpyDeviceToHost);

        // Modificar as variáveis no host com base nos resultados do kernel
        for (int i = 0; i < size; ++i) {
            a[i] += 1;  // Incrementar 1 em a com base nos resultados do kernel
        }

        // Copiar as variáveis modificadas de volta para o device (se necessário)
        hipMemcpy(d_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    }

    // Exibir o resultado final (aqui você pode realizar operações adicionais se necessário)
    for (int i = 0; i < size; i++)
    {
        std::cout << a[i] << " ";
    }
    std::cout << "\n";
    for (int i = 0; i < size; i++)
    {
        std::cout << b[i] << " ";
    }
    std::cout << "\n";
    std::cout << "Resultado da soma final:\n";
    for (int i = 0; i < size; ++i) {
        std::cout << c[i] << " ";
    }
    std::cout << "\n";
    // Liberar memória no device
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
